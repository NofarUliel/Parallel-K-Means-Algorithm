#include "hip/hip_runtime.h"
#include "kernel.h"
__global__ void updatePointsByTime(Point* allPoints, Point* result, int numOfPoints, double time)
{
	int index = (blockIdx.x*blockDim.x) + threadIdx.x;
	if (index < numOfPoints) {
		result[index].x = allPoints[index].orgx + allPoints[index].vx * time;
		result[index].y = allPoints[index].orgy + allPoints[index].vy * time;
		result[index].z = allPoints[index].orgz + allPoints[index].vz * time;

	}

}

// Helper function for using CUDA to add vectors in parallel.
hipError_t updatePointByTime_CUDA(Point* allPoints, Point* result, int numOfPoints, double time)
{
	Point *dev_points = 0;
	Point *dev_result = 0;
	hipError_t cudaStatus;
	int numOfblock = numOfPoints / MAX_BLOCK_SIZE_CUDA;
	if (numOfPoints%MAX_BLOCK_SIZE_CUDA != 0)
		numOfblock = numOfPoints / MAX_BLOCK_SIZE_CUDA + 1;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	// Allocate GPU buffers for three vectors (one input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_points, numOfPoints * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&dev_result, numOfPoints * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_points, allPoints, numOfPoints * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_result, result, numOfPoints * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	// Launch a kernel on the GPU with one thread for each element.
	updatePointsByTime << <numOfblock, MAX_BLOCK_SIZE_CUDA >> >(dev_points, dev_result, numOfPoints, time);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(result, dev_result, numOfPoints * sizeof(Point), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_result);
	hipFree(dev_points);

	return cudaStatus;
}